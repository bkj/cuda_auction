#include "hip/hip_runtime.h"
#ifndef MAIN_AUCTION
#define MAIN_AUCTION

#include <iostream>
#include <string>
#include <fstream>

#include <stdio.h>
#include <stdlib.h>
// #include "nodet_auction_kernel.cu" // Faster, but non deterministic
#include "auction_kernel.cu"

// --
// Define constants


#ifndef __RUN_VARS
#define __RUN_VARS
#define MAX_NODES       20000 // Dimension of problem
#define BLOCKSIZE       32 // How best to set this?
#define AUCTION_MAX_EPS 1.0 // Larger values mean solution is more approximate
#define AUCTION_MIN_EPS 1.0 / 4.0
#define AUCTION_FACTOR  1.0 / 2.0
#define NUM_RUNS        3
#endif

int load_data(float *raw_data) {
    std::ifstream input_file("graph", std::ios_base::in);
    
    std::cerr << "load_data: start" << std::endl;
    int i = 0;
    float val;
    while(input_file >> val) {
        raw_data[i] = val;
        i++;
        if(i > MAX_NODES * MAX_NODES) {
            std::cerr << "load_data: ERROR -- data file too large" << std::endl;
            return -1;
        }
    }
    std::cerr << "load_data: finish" << std::endl;
    return (int)sqrt(i);
}

extern "C" {

int run_auction(float* h_data, int* h_person2item, int num_nodes, 
    int verbose, int num_runs, float auction_max_eps, float auction_min_eps, float auction_factor){
    
    int h_numAssign;
    
    float* d_data;
    float* d_bids;
    float* d_prices;
    int* d_bidders;
    int* d_sbids;
    int* d_person2item;
    int* d_item2person;
    
    //using atomic operations, counts the number of assigns, 
    //otherwise, used as a boolean that is set whenever there is an unassigned person
    int* d_numAssign = 0;

    hipMalloc((void **)&d_data,        num_nodes * num_nodes * sizeof(float));
    hipMalloc((void **)&d_bids,        num_nodes * num_nodes * sizeof(float));
    hipMalloc((void **)&d_prices,      num_nodes             * sizeof(float));
    hipMalloc((void **)&d_bidders,     num_nodes * num_nodes * sizeof(int));
    hipMalloc((void **)&d_sbids,       num_nodes * sizeof(int));
    hipMalloc((void **)&d_person2item, num_nodes * sizeof(int));
    hipMalloc((void **)&d_item2person, num_nodes * sizeof(int));
    hipMalloc((void **)&d_numAssign,           1 * sizeof(int)) ;
    
    hipMemcpy(d_data, h_data, sizeof(float) * num_nodes * num_nodes, hipMemcpyHostToDevice);

    dim3 dimBlock(BLOCKSIZE, 1, 1);
    int gx = ceil(num_nodes / (double) dimBlock.x);
    dim3 dimGrid(gx, 1, 1);
    
    for(int run_num = 0; run_num < num_runs; run_num++) {
        
        // Start timer
        hipEvent_t start, stop;
        float milliseconds = 0;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        // Reset data structures
        hipMemset(d_prices, 0.0, num_nodes * sizeof(float));
        hipDeviceSynchronize();

        float auction_eps = auction_max_eps;
        while(auction_eps >= auction_min_eps) {
            h_numAssign = 0;
            hipMemset(d_bidders,        0, num_nodes * num_nodes * sizeof(int));
            hipMemset(d_person2item,   -1, num_nodes             * sizeof(int));
            hipMemset(d_item2person,   -1, num_nodes             * sizeof(int));
            hipMemset(d_numAssign,      0, 1                     * sizeof(int));
            hipDeviceSynchronize();
            
            while(h_numAssign < num_nodes){
                          
                hipMemset(d_bids,  0, num_nodes * num_nodes * sizeof(float));
                hipMemset(d_sbids, 0, num_nodes             * sizeof(int));
                hipDeviceSynchronize();
                            
                run_bidding<<<dimBlock, dimGrid>>>(
                    num_nodes,
                    d_data,
                    d_person2item,
                    d_bids,
                    d_bidders,
                    d_sbids,
                    d_prices,
                    auction_eps
                );
                run_assignment<<<dimBlock, dimGrid>>>(
                    num_nodes,
                    d_person2item,
                    d_item2person,
                    d_bids,
                    d_bidders,
                    d_sbids,
                    d_prices,
                    d_numAssign
                );
                hipDeviceSynchronize();
                
                hipMemcpy(&h_numAssign, d_numAssign, sizeof(int) * 1, hipMemcpyDeviceToHost);
            }
            
            auction_eps *= auction_factor;
        }
        
        // Stop timer
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        if(verbose) {
            std::cerr << 
                "run_num="         << run_num      << 
                " | h_numAssign="  << h_numAssign  <<
                " | milliseconds=" << milliseconds << std::endl;            
        }
        
        hipDeviceSynchronize();
     }
     
    // Read out results
    hipMemcpy(h_person2item, d_person2item, sizeof(int) * num_nodes, hipMemcpyDeviceToHost);
    
    hipFree(d_data);
    hipFree(d_bids);
    hipFree(d_prices);  
    hipFree(d_person2item); 
    hipFree(d_item2person); 
    hipFree(d_numAssign);
        
    return 0;
}    
}


void init_device() {
    // Init devices        
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    if(deviceCount<1){
        printf("There is no device detected.\n");
        exit(1);
    }
    int device=0;
    hipDeviceProp_t deviceProp;
    for (device = 0; device < deviceCount; ++device) {        
        if(hipGetDeviceProperties(&deviceProp, device) == hipSuccess) {
            if(deviceProp.major >= 1) {
                break;
            }
        }
    }
    if(device == deviceCount) {
        printf("There is no device supporting CUDA.\n");
        exit(1);
    }    
    hipSetDevice(device);
}


int main(int argc, char **argv)
{
    init_device();

    // Load data
    float* raw_data = (float *)malloc(sizeof(float) * MAX_NODES * MAX_NODES);
    int num_nodes = load_data(raw_data);
    if(num_nodes <= 0) {
        return 1;
    }
    
    float* h_data      = (float *)realloc(raw_data, sizeof(float) * num_nodes * num_nodes);
    int* h_person2item = (int *)malloc(sizeof(int) * num_nodes);
    int verbose        = 1;
    run_auction(h_data, h_person2item, num_nodes, 
        verbose, NUM_RUNS, AUCTION_MAX_EPS, AUCTION_MIN_EPS, AUCTION_FACTOR);

    // Print results
    float score = 0;
    for (int i = 0; i < num_nodes; i++) {
        std::cout << i << " " << h_person2item[i] << std::endl;
        score += h_data[i + num_nodes * h_person2item[i]];
    }
    
    std::cerr << "score=" << (int)score << std::endl;        

    free(h_data);
    free(h_person2item);
}

#endif