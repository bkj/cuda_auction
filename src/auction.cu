#include "hip/hip_runtime.h"
// auction.cu
//
// !! For best performance, I think the datalayout
// needs to be transposed.  Eg:
//      i + num_nodes * j
// instead of the current
//      i * num_nodes + j

#ifndef MAIN_AUCTION
#define MAIN_AUCTION

#include <iostream>
#include <string>
#include <fstream>
#include <chrono>

#include <stdio.h>
#include <stdlib.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

// --
// Define constants


#ifndef __RUN_VARS
#define __RUN_VARS
#define MAX_NODES       20000 // Dimension of problem
#define BLOCKSIZE       32 // How best to set this?
#define AUCTION_MAX_EPS 1.0 // Larger values mean solution is more approximate
#define AUCTION_MIN_EPS 1.0
#define AUCTION_FACTOR  0.0
#define NUM_RUNS        10

// Uncomment to run dense version
// #define DENSE
#endif

#include "topdot.cpp"
#ifdef DENSE
    #include "auction_kernel_dense.cu"
#else
    #include "auction_kernel_csr.cu"
#endif


int load_data(float *raw_data) {
    std::ifstream input_file("graph", std::ios_base::in);

    std::cerr << "load_data: start" << std::endl;
    int i = 0;
    float val;
    while(input_file >> val) {
        raw_data[i] = val;
        i++;
        if(i > MAX_NODES * MAX_NODES) {
            std::cerr << "load_data: ERROR -- data file too large" << std::endl;
            return -1;
        }
    }
    std::cerr << "load_data: finish" << std::endl;
    return (int)sqrt(i);
}

extern "C" {

int run_auction(
    int    num_nodes,
    int    num_edges,

    float* h_data,      // data
    int*   h_offsets,   // offsets for items
    int*   h_columns,

    int*   h_person2item, // results

    float auction_max_eps,
    float auction_min_eps,
    float auction_factor,

    int num_runs,
    int verbose
)
{
    // --
    // CUDA options

    dim3 dimBlock(BLOCKSIZE, 1, 1);
    int gx = ceil(num_nodes / (double) dimBlock.x);
    dim3 dimGrid(gx, 1, 1);

    // --
    // Declare variables

    float* d_data;
    int*   d_offsets;
    int*   d_columns;

    int* d_person2item;
    int* d_item2person;

    float* d_bids;
    float* d_prices;
    int*   d_bidders; // unused
    int*   d_sbids;

    int  h_numAssign;
    int* d_numAssign = 0;
    float* d_rand;

    // --
    // Allocate device memory
    hipMalloc((void **)&d_data,    num_edges * sizeof(float));
    hipMalloc((void **)&d_columns, num_edges * sizeof(float));
    hipMalloc((void **)&d_offsets, (num_nodes + 1) * sizeof(int));

    hipMalloc((void **)&d_person2item, num_nodes * sizeof(int));
    hipMalloc((void **)&d_item2person, num_nodes * sizeof(int));

    hipMalloc((void **)&d_bids,    num_nodes * num_nodes * sizeof(float));
    hipMalloc((void **)&d_prices,  num_nodes * sizeof(float));
    hipMalloc((void **)&d_bidders, num_nodes * num_nodes * sizeof(int)); // unused
    hipMalloc((void **)&d_sbids,   num_nodes * sizeof(int));

    hipMalloc((void **)&d_numAssign, 1 * sizeof(int)) ;
    hipMalloc((void **)&d_rand,      num_nodes * num_nodes * sizeof(float)) ;

    // --
    // Copy from host to device

    hipMemcpy(d_data,    h_data,    num_edges       * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_columns, h_columns, num_edges       * sizeof(int),   hipMemcpyHostToDevice);
    hipMemcpy(d_offsets, h_offsets, (num_nodes + 1) * sizeof(int),   hipMemcpyHostToDevice);

    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, 123);
    hiprandGenerateUniform(gen, d_rand, num_nodes * num_nodes);

    for(int run_num = 0; run_num < num_runs; run_num++) {

        hipMemset(d_prices, 0.0, num_nodes * sizeof(float));

        // Start timer
        hipEvent_t start, stop;
        float milliseconds = 0;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        float auction_eps = auction_max_eps;
        while(auction_eps >= auction_min_eps) {
            h_numAssign = 0;
            hipMemset(d_bidders,        0, num_nodes * num_nodes * sizeof(int)); // unused
            hipMemset(d_person2item,   -1, num_nodes * sizeof(int));
            hipMemset(d_item2person,   -1, num_nodes * sizeof(int));
            hipMemset(d_numAssign,      0, 1         * sizeof(int));
            hipDeviceSynchronize();

            int counter = 0;
            while(h_numAssign < num_nodes){
                counter += 1;
                hipMemset(d_bids,  0, num_nodes * num_nodes * sizeof(float));
                hipMemset(d_sbids, 0, num_nodes * sizeof(int));
                hipDeviceSynchronize();

                run_bidding<<<dimBlock, dimGrid>>>(
                    num_nodes,

                    d_data,
                    d_offsets,
                    d_columns,

                    d_person2item,
                    d_bids,
                    d_bidders,
                    d_sbids,
                    d_prices,
                    auction_eps,
                    d_rand
                );
                run_assignment<<<dimBlock, dimGrid>>>(
                    num_nodes,
                    d_person2item,
                    d_item2person,
                    d_bids,
                    d_bidders,
                    d_sbids,
                    d_prices,
                    d_numAssign
                );
                hipDeviceSynchronize();

                hipMemcpy(&h_numAssign, d_numAssign, sizeof(int) * 1, hipMemcpyDeviceToHost);
                // std::cerr << "h_numAssign=" << h_numAssign << std::endl;
            }
            if(verbose) {
                std::cerr << "counter=" << counter << std::endl;
            }

            auction_eps *= auction_factor;
        }
        hipDeviceSynchronize();

        // Stop timer
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        if(verbose) {
            std::cerr <<
                "run_num="         << run_num      <<
                " | h_numAssign="  << h_numAssign  <<
                " | milliseconds=" << milliseconds << std::endl;
        }

        hipDeviceSynchronize();
     }

    // Read out results
    hipMemcpy(h_person2item, d_person2item, sizeof(int) * num_nodes, hipMemcpyDeviceToHost);

    hipFree(d_data);
    hipFree(d_columns);
    hipFree(d_offsets);
    hipFree(d_person2item);
    hipFree(d_item2person);
    hipFree(d_bids);
    hipFree(d_prices);
    hipFree(d_bidders);
    hipFree(d_sbids);
    hipFree(d_numAssign);
    hipFree(d_rand);

    return 0;
} // end run_auction

int run_auction_python(
    int    num_nodes,
    int    num_edges,

    float* h_data,      // data
    int*   h_offsets,   // offsets for items
    int*   h_columns,

    int*   h_person2item, // results

    float auction_max_eps,
    float auction_min_eps,
    float auction_factor,

    int num_runs,
    int verbose
) {

    hipEvent_t auction_start, auction_stop;
    float milliseconds = 0;
    hipEventCreate(&auction_start);
    hipEventCreate(&auction_stop);
    hipEventRecord(auction_start, 0);

    run_auction(
        num_nodes,
        num_edges,

        h_data,      // data
        h_offsets,   // offsets for items
        h_columns,

        h_person2item, // results

        auction_max_eps,
        auction_min_eps,
        auction_factor,

        num_runs,
        0
    );
    hipEventRecord(auction_stop, 0);
    hipEventSynchronize(auction_stop);
    hipEventElapsedTime(&milliseconds, auction_start, auction_stop);
    hipEventDestroy(auction_start);
    hipEventDestroy(auction_stop);
    if(verbose > 0) {
        std::cerr << "run_auction     " << milliseconds << std::endl;
    }
    return 0;
}

int dot_auction(
        int num_nodes,
        int *Ap, int *Aj, double *Ax,
        int *Bp, int *Bj, double *Bx,
        int k,
        int *h_person2item,
        int verbose
) {

    std::chrono::high_resolution_clock::time_point topdot_start = std::chrono::high_resolution_clock::now();

    int* h_columns   = (int *)malloc(sizeof(int) * num_nodes * k);
    double* h_data_d = (double *)malloc(sizeof(double) * num_nodes * k);
    float* h_data    = (float *)malloc(sizeof(float) * num_nodes * k);
    int* h_offsets   = (int *)malloc(sizeof(int) * num_nodes + 1);
    _topdot(num_nodes, num_nodes, Ap, Aj, Ax, Bp, Bj, Bx, k, -1, h_columns, h_data_d);
    h_offsets[0] = 0;
    for(int i = 1; i < num_nodes + 1; i++) {
        h_offsets[i] = i * k;
    }

    for(int i = 0; i < num_nodes * k; i++) {
        h_data[i] = (float)h_data_d[i];
        if(verbose > 1) {
            std::cerr << h_columns[i] << ":" << h_data[i] << " ";
            if((i + 1) % k == 0) {
                std::cerr << std::endl;
            }
        }
    }
    free(h_data_d);

        // Stop timer
    std::chrono::high_resolution_clock::time_point topdot_stop = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> time_span =
        std::chrono::duration_cast<std::chrono::duration<double>>(topdot_stop - topdot_start);

    if(verbose > 0) {
        std::cerr << "topdot          " << 1000 * time_span.count() << std::endl;
    }

    // --
    // Auction algorithm

    hipEvent_t auction_start, auction_stop;
    float milliseconds = 0;
    hipEventCreate(&auction_start);
    hipEventCreate(&auction_stop);
    hipEventRecord(auction_start, 0);

    run_auction(
        (int)num_nodes,
        (int)num_nodes * k,

        h_data,
        h_offsets,
        h_columns,

        h_person2item,

        (float)1.0,
        (float)1.0,
        (float)0.0,

        (int)1, // 1 run
        (int)0  // not verbose
    );

    hipEventRecord(auction_stop, 0);
    hipEventSynchronize(auction_stop);
    hipEventElapsedTime(&milliseconds, auction_start, auction_stop);
    hipEventDestroy(auction_start);
    hipEventDestroy(auction_stop);
    if(verbose > 0) {
        std::cerr << "run_auction     " << milliseconds << std::endl;
    }

    free(h_columns);
    free(h_data);
    free(h_offsets);

    return 0;
} // end dot_auction

} // end extern


void init_device() {
    // Init devices
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    if(deviceCount<1){
        printf("There is no device detected.\n");
        exit(1);
    }
    int device=0;
    hipDeviceProp_t deviceProp;
    for (device = 0; device < deviceCount; ++device) {
        if(hipGetDeviceProperties(&deviceProp, device) == hipSuccess) {
            if(deviceProp.major >= 1) {
                break;
            }
        }
    }
    if(device == deviceCount) {
        printf("There is no device supporting CUDA.\n");
        exit(1);
    }
    hipSetDevice(device);
}


int main(int argc, char **argv)
{
#ifdef DENSE
    std::cerr << "auction_kernel_dense.cu" << std::endl;
#else
    std::cerr << "auction_kernel_csr.cu" << std::endl;
#endif

    init_device();

    // Load data
    float* raw_data = (float *)malloc(sizeof(float) * MAX_NODES * MAX_NODES);
    int num_nodes = load_data(raw_data);
    int num_edges = num_nodes * num_nodes;
    if(num_nodes <= 0) {
        return 1;
    }

    float* h_data  = (float *)realloc(raw_data, sizeof(float) * num_nodes * num_nodes);

    // Dense
    int* h_offsets = (int *)malloc(sizeof(int) * num_nodes + 1);
    h_offsets[0] = 0;
    for(int i = 1; i < num_nodes + 1; i++) {
        h_offsets[i] = i * num_nodes;
    }

    int* h_columns = (int *)malloc(sizeof(int) * num_edges);
    for(int i = 0; i < num_edges; i++) {
        h_columns[i] = i % num_nodes;
    }

    int* h_person2item = (int *)malloc(sizeof(int) * num_nodes);

    int verbose = 1;

    run_auction(
        num_nodes,
        num_edges,

        h_data,
        h_offsets,
        h_columns,

        h_person2item,

        AUCTION_MAX_EPS,
        AUCTION_MIN_EPS,
        AUCTION_FACTOR,

        NUM_RUNS,
        verbose
    );

    // Print results
    float score = 0;
    for (int i = 0; i < num_nodes; i++) {
        std::cout << i << " " << h_person2item[i] << std::endl;
        // score += h_data[i + num_nodes * h_person2item[i]];
        score += h_data[i * num_nodes + h_person2item[i]];
    }

    std::cerr << "score=" << (int)score << std::endl;

    free(h_data);
    free(h_offsets);
    free(h_columns);
    free(h_person2item);
}

#endif
