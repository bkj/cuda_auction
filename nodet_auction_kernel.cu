
#include <hip/hip_runtime.h>
__global__ void run_bidding(
    const int num_nodes, float *data, int *person2item, float *bids, int *bidders, int *sbids, float *prices, float auction_eps){
    
    int i = blockDim.x * blockIdx.x + threadIdx.x; // person index
    if(i < num_nodes){
        if(person2item[i] == -1) {
            
            int fir_maxObj        = -1;
            float fir_maxObjValue = 0;
            float sec_maxObjValue = -1000;
            float temp_ObjValue   = 0;
            
            fir_maxObj      = 0;
            fir_maxObjValue = data[i] - prices[0];
            
            for(int j = 1; j < num_nodes; j++){
                temp_ObjValue = data[i + num_nodes * j] - prices[j];
                if(temp_ObjValue > fir_maxObjValue){
                    sec_maxObjValue = fir_maxObjValue;
                    
                    fir_maxObj      = j;
                    fir_maxObjValue = temp_ObjValue;
                } else if(temp_ObjValue > sec_maxObjValue){
                    sec_maxObjValue = temp_ObjValue;
                }        
            }
            
            float bid = data[i + num_nodes * fir_maxObj] - sec_maxObjValue + auction_eps;
            int idx = atomicAdd(sbids + fir_maxObj, 1);
            bids[idx + num_nodes * fir_maxObj] = bid;
            bidders[idx + num_nodes * fir_maxObj] = i;
        }
    }
}


__global__ void run_assignment(
    const int num_nodes, int *person2item, int *item2person, float *bids, int *bidders, int *sbids, float *prices, int *num_assigned){
    
    int j = blockDim.x * blockIdx.x + threadIdx.x; // item index
    if(j < num_nodes) {
        int num_bidders = sbids[j];
        if(num_bidders != 0) {
            float high_bid  = bids[0 + num_nodes * j];
            int high_bidder = bidders[0 + num_nodes * j];
            
            float tmp_bid = -1.0;
            for(int i = 1; i < num_bidders; i++){
                tmp_bid = bids[i + num_nodes * j];
                if(tmp_bid > high_bid){
                    high_bid    = tmp_bid;
                    high_bidder = bidders[i + num_nodes * j];
                }
            }
            
            int current_person = item2person[j];
            if(current_person >= 0){
                person2item[current_person] = -1; 
            } else {
                atomicAdd(num_assigned, 1);
            }
            
            prices[j]                = high_bid;
            person2item[high_bidder] = j;
            item2person[j]           = high_bidder;
        }
    }
}
