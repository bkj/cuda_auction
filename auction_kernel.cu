
#include <hip/hip_runtime.h>
#include <assert.h>

__global__ void run_bidding(
    const int num_nodes, int *data, int *person2item, int *bids, int *sbids, int *prices, int auction_eps){
    
    int i = blockDim.x * blockIdx.x + threadIdx.x; // person index
    if(i < num_nodes){
        if(person2item[i] == -1) {

            int fir_maxObj      = -1;
            int fir_maxObjValue = 0;
            int sec_maxObjValue = -1000;
            int temp_ObjValue   = 0;

            fir_maxObj      = 0;
            fir_maxObjValue = data[i] - prices[0];
            
            for(int j = 1; j < num_nodes; j++){
                temp_ObjValue = data[i + num_nodes * j] - prices[j];
                if(temp_ObjValue > fir_maxObjValue){
                    sec_maxObjValue = fir_maxObjValue;
                    
                    fir_maxObj      = j;
                    fir_maxObjValue = temp_ObjValue;
                } else if(temp_ObjValue > sec_maxObjValue){
                    sec_maxObjValue = temp_ObjValue;
                }        
            }
            
            int bid = data[i + num_nodes * fir_maxObj] - sec_maxObjValue + auction_eps;
            bids[i + num_nodes * fir_maxObj] = bid;
            atomicMax(sbids + fir_maxObj, bid);
        }
    }
}


__global__ void run_assignment(
    const int num_nodes, int *person2item, int *item2person, int *bids, int *sbids, int *prices, int *num_assigned){
    
    int j = blockDim.x * blockIdx.x + threadIdx.x; // item index
    if(j < num_nodes) {
        if(sbids[j] != 0) {
            int high_bid    = 0.0;
            int high_bidder = -1;
            
            int tmp_bid = -1;
            for(int i = 0; i < num_nodes; i++){        
                tmp_bid = bids[i + num_nodes * j]; 
                if(tmp_bid > high_bid){
                    high_bid    = tmp_bid;
                    high_bidder = i;
                }
            }
            
            int current_person = item2person[j];
            if(current_person >= 0){
                person2item[current_person] = -1; 
            } else {
                atomicAdd(num_assigned, 1);                
            }
            
            prices[j]                = high_bid;
            person2item[high_bidder] = j;
            item2person[j]           = high_bidder;
        }
    }
}
